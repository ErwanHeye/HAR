#include "hip/hip_runtime.h"
__device__ double computeMatch(unsigned char *im,
		    int im_step,
		    unsigned char *bl,
		    int bl_step,
		    int bl_cols,
		    int bl_rows,
		    int oi, 
		    int oj, 
		    int stride){
  
  if (!im || !bl) return 0.0;

  double nb = (bl_cols*bl_rows);
  double x = 0;
  for(int i = 0;i < bl_rows-stride+1;i+= stride){
    for(int j = 0;j < bl_cols-stride+1;j+= stride){
      unsigned char v1 = im[INDXs(im_step,oi+i,oj+j)];
      unsigned char v2 = bl[INDXs(bl_step,i,j)];
      x += (v2-v1)*(v2-v1);
      //im[INDXs(im_step,oi+i,oj+j)] = ABS(v2-v1);
    }
  }
  x = x / nb;
  //  printf("%f\n",x);
  return x;
}

__global__ void blockMatching_kernel(
	int jend,int stride,int im, int im_step,
	int bl_step,int bl_cols,int bl_rows){
		
		int tid = blockDim.x * blockIdx.x + threadIdx.x;
		for(int j = 0;j < jend-stride+1;j+=stride){
		  double x = computeMatch(im,im_step,
					  bl,bl_step,bl_cols,bl_rows,
					  tid,j,stride);
		  
		  if(x < minVal){
			minVal = x;
			coord_i_min = tid;
			coord_j_min = j;
			bestScale = s;
			bestRotation = r;

		  }
		}
}
