#include "hip/hip_runtime.h"
__device__ double computeMatch(unsigned char *im,
		    int im_step,
		    unsigned char *bl,
		    int bl_step,
		    int bl_cols,
		    int bl_rows,
		    int oi, 
		    int oj, 
		    int stride){
  
  if (!im || !bl) return 0.0;

  double nb = (bl_cols*bl_rows);
  double x = 0;
  for(int i = 0;i < bl_rows-stride+1;i+= stride){
    for(int j = 0;j < bl_cols-stride+1;j+= stride){
      unsigned char v1 = im[INDXs(im_step,oi+i,oj+j)];
      unsigned char v2 = bl[INDXs(bl_step,i,j)];
      x += (v2-v1)*(v2-v1);
      //im[INDXs(im_step,oi+i,oj+j)] = ABS(v2-v1);
    }
  }
  x = x / nb;
  //  printf("%f\n",x);
  return x;
}

struct DataOut{
	double minVal;
	int coord_i_min;
	int coord_j_min;
};

__global__ void blockMatching_kernel(int jend,int stride,unsigned char* im, int im_step, unsigned char *bl, int bl_step,int bl_cols,int bl_rows, DataOut* result){
		
	__shared__ DataOut tab_data_out[blockDim.x * blockDim.y];
	
	DataOut temp;
	temp.minVal=DBL_MAX;	
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	for(int j = 0;j < jend-stride+1;j+=stride){
	  double x = computeMatch(im,im_step,
				  bl,bl_step,bl_cols,bl_rows,
				  tid,j,stride);
	  
		 if(x<temp.minVal){
			 temp.minVal=x;
			 temp.coord_i_min=tid;
			 temp.coord_j_min=j;
		}
	}
			
	tab_data_out[tid]=temp;
	__syncthreads();
	if(threadIdx.x==0 && threadIdx.y == 0){
		for(int i = 1; i < blockDim.x * blockDim.y; i++){
			if(temp.minVal >= tab_data_out[i].minVal){
				temp = tab_data_out[i];
			}
		}
		result = &temp;
	}
	 
}
	
